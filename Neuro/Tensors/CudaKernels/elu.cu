
#include <hip/hip_runtime.h>
﻿extern "C" {
    __global__ void Run(int n, float* __restrict input, float* __restrict result, float alpha)
    {
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		if (i < n)
            result[i] = input[i] > 0 ? input[i] : alpha * (exp(input[i]) - 1);
	}
}