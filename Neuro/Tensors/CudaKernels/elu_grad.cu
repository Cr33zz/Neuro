
#include <hip/hip_runtime.h>
﻿extern "C" {
    __global__ void Run(int n, float* __restrict output, float* __restrict outputGradient, float* __restrict result, float alpha)
    {
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		if (i < n)
            result[i] = (output[i] > 0 ? 1 : (output[i] + alpha)) * outputGradient[i];
	}
}